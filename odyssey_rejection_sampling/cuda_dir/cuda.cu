#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <sys/time.h>
#include <math.h>

__global__ void kernel(int* count_d, float* randomnums)
{
	int i;
	double x,y,z;
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	i = tid;
	int xidx = 0, yidx = 0;

	xidx = (i+i);
	yidx = (xidx+1);

	x = randomnums[xidx];
	y = randomnums[yidx];
	z = 1/sqrt(2*M_PI) * exp(-0.5*pow(x,2));

	if (y<=z)
		count_d[tid] = 1;
	else
		count_d[tid] = 0;	
}

void CUDAErrorCheck()
{
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{	
		printf("CUDA error : %s (%d)\n", hipGetErrorString(error), error);
		exit(0);
	}
}

int main(int argc,char* argv[])
{
	int niter = atoi(argv[1]);
        int repetitions = 3;
        int j = 0;

        for (j=0; j<repetitions; j++) 
        {
	   float *randomnums;
	   double phi;
	   hipMalloc((void**)&randomnums, (2*niter)*sizeof(float));
	   // Use CuRand to generate an array of random numbers on the device
	   int status;
	   hiprandGenerator_t gen;
	   status = hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MRG32K3A);
	   status |= hiprandSetPseudoRandomGeneratorSeed(gen, 2138+j);
	   // status |= curandSetPseudoRandomGeneratorSeed(gen, 4294967296ULL^time(NULL));
	   status |= hiprandGenerateUniform(gen, randomnums, (2*niter));
	   status |= hiprandDestroyGenerator(gen);
	   if (status != HIPRAND_STATUS_SUCCESS)
	   {
		printf("CuRand Failure\n");
		exit(EXIT_FAILURE);
	   }

	   int threads = 1000;
	   int blocks = 100;
	   int* count_d;
	   int *count = (int*)malloc(blocks*threads*sizeof(int));
	   unsigned int reducedcount = 0;
	   hipMalloc((void**)&count_d, (blocks*threads)*sizeof(int));
	   CUDAErrorCheck();

struct timeval begin, end;
gettimeofday(&begin, NULL);

           hipEvent_t start, stop;
           hipEventCreate(&start);
           hipEventCreate(&stop);
           hipEventRecord(start, 0); 
	   //one point per thread
	   kernel <<<blocks, threads>>> (count_d, randomnums);
	   hipDeviceSynchronize();
	   CUDAErrorCheck();
	   hipMemcpy(count, count_d, blocks*threads*sizeof(int), hipMemcpyDeviceToHost);
	   int i = 0;
	   //reduce array into int
	   for(i = 0; i<niter; i++)
		reducedcount += count[i];
           hipEventRecord(stop, 0); 
           float elapsedTime = 0;
           hipEventElapsedTime(&elapsedTime, start, stop);

gettimeofday(&end, NULL);
double elapsed = (end.tv_sec - begin.tv_sec) + ((end.tv_usec - begin.tv_usec)/1000000.0);

	   hipFree(randomnums);
	   hipFree(count_d);
	   free(count);
           hipEventDestroy(start);
           hipEventDestroy(stop);

	   phi = ((double)reducedcount/niter)*1.0 + 0.5;
	   printf("CUDA - area to left of 1 on standard normal: %f\n", phi);
           //printf("runtime: %f\n", elapsedTime);
           printf("runtime: %f\n", elapsed);
           //printf("runtime: %f\n", seconds);
       }
  
       return 0;
        
}
